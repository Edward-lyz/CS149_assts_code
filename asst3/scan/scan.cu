#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/functional.h>
#include <thrust/transform.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256


// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result

__global__ void hs_scan_kernel(int* input, int N, int k) {
    // get the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // check the thread index is within the array bounds
    if (i < N) {
        // get the distance to the previous element
        int offset = 1 << k;
        // check the previous element exists
        if (i >= offset) {
            // perform the scan operation
            input[i] = input[i] + input[i - offset];
        }
    }
}

__global__ void shift_right_kernel(int* result, int N, int M) {
    // get the thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // check the boundary
    if (i < M) {
        // if the index is within N, copy the previous element
        if (i < N) {
            result[i] = result[i-1];
        }
        // if the index is equal to 0, set it to 0
        else if (i == 0) {
            result[i] = 0;
        }
        // otherwise, do nothing
    }
}

void exclusive_scan(int* input, int N, int* result)
{   
    // printf("start exclusive scan");s
    // copy the input array to the result array
    hipMemcpy(result, input, N * sizeof(int), hipMemcpyDeviceToDevice);
    // get the next power of 2 of N
    int M = nextPow2(N);
    // set the block and grid dimensions
    int blockSize = 256;
    int gridSize = (M + blockSize - 1) / blockSize;
    // loop over the log2(M) steps
    for (int k = 0; k <= log2(M); k++) {
        // call the scan kernel
        hs_scan_kernel<<<gridSize, blockSize>>>(result, N, k);
        // synchronize the device
        hipDeviceSynchronize();
    }
    // call the shift right kernel with the same grid and block dimensions
    shift_right_kernel<<<gridSize, blockSize>>>(result, N, M);
    // synchronize the device
    hipDeviceSynchronize();
    // copy the result array back to the host memory
    // hipMemcpy(result+4, result, (N-1) * sizeof(int), hipMemcpyDeviceToDevice);
}

//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input;
    int N = end - inarray;  

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);
    // printf("the rounded_lenth is: %d",rounded_length);
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, N, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
       
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
   
    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}

// A kernel function to convert the input array to a flag array
__global__ void flag_kernel(int* device_input, int* device_flag, int length) {
    // Get the thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Check the boundary
    if (index < length - 1) {
        // Compare the current element with the next element
        if (device_input[index] == device_input[index + 1]) {
            // Set the flag to 1 if they are equal
            device_flag[index] = 1;
        } else {
            // Set the flag to 0 otherwise
            device_flag[index] = 0;
        }
    }
    // The last element has no next element, so set the flag to 0
    if (index == length - 1) {
        device_flag[index] = 0;
    }
    // Synchronize the threads
    __syncthreads();
}

// A kernel function to convert the scan array to the output array
__global__ void output_kernel(int* device_scan, int* device_output, int length) {
    // Get the thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Check the boundary
    if (index < length) {
        // Use a shared variable to store the current position of the output array
        __shared__ int pos;
        // Initialize the position to 0 in the first thread
        if (threadIdx.x == 0) {
            pos = 0;
        }
        // Synchronize the threads
        __syncthreads();
        // Check if the current element is a flag
        if (device_scan[index] > device_scan[index - 1]) {
            // Get the current position of the output array atomically
            int p = atomicAdd(&pos, 1);
            // Store the index of the flag to the output array
            device_output[p] = index;
        }
        // Synchronize the threads
        __syncthreads();
    }
}

// A function to find the repeats using exclusive_scan
int find_repeats(int* device_input, int length, int* device_output) {
    // Allocate memory for the flag array and the scan array
    int* device_flag;
    int* device_scan;
    hipMalloc(&device_flag, length * sizeof(int));
    hipMalloc(&device_scan, length * sizeof(int));
    // Define the block size and the grid size
    int block_size = 256;
    int grid_size = (length + block_size - 1) / block_size;
    // Launch the flag kernel
    flag_kernel<<<grid_size, block_size>>>(device_input, device_flag, length);
    // Launch the exclusive_scan function
    exclusive_scan(device_flag, length, device_scan);
    // Launch the output kernel
    output_kernel<<<grid_size, block_size>>>(device_scan, device_output, length);
    // Copy the last element of the scan array to the host memory
    int result;
    hipMemcpy(&result, device_scan + length - 1, sizeof(int), hipMemcpyDeviceToHost);
    // Free the allocated memory
    hipFree(device_flag);
    hipFree(device_scan);
    // Return the number of pairs found
    return result;
}

//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);
    // due to GPU's paralle computing, the result needs to be sorted
    std::sort(output,output+length);
    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime; 
    return duration;
}



void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
